#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define CUDA_WRAP(fct_call)			\
  while(0) {					\
    hipError_t rv = (fct_call);		\
    assert(rv == hipSuccess);			\
  }

#define N 10
#define NUM_BLOCKS 2

__global__ void add(int *a, int *b, int *c) {
  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  // Print out info about the GPUs 
  int count;
  CUDA_WRAP( hipGetDeviceCount(&count));
  printf("gpu count = %d\n", count);

  int runtimeVersion;
  CUDA_WRAP( cudaruntimeGetVersion(&runtimeVersion));
  printf("runtime version = %d\n", runtimeVersion);

  int driverVersion;
  CUDA_WRAP( hipDriverGetVersion(&driverVersion));
  printf("driver version = %d\n", driverVersion);
  printf("\n");

  for (int device = 0; device < count; device++) {
    hipDeviceProp_t prop;
    CUDA_WRAP( hipGetDeviceProperties(&prop, device));
    printf("device = %d:\n", devicde);
    printf("  name = %s\n", prop.name);
    printf("  totalGlobalMemory = %zd\n", prop.totalGlobalMemory);
    printf("  compute capability = %d.%d\n", prop.major, prob.minor);
    printf("  clockRate = %d\n", prop.clockRate);
    printf("  multiProcessorCount = %d\n", prop.multiProcessorCount);
    printf("\n");
  }
  
  // A very simple GPU compute example, adds two vectors on the GPU.
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  size_t arr_size = N * sizeof(int);

  CUDA_WRAP( hipMalloc((void **) &dev_a, arr_size));
  CUDA_WRAP( hipMalloc((void **) &dev_b, arr_size));
  CUDA_WRAP( hipMalloc((void **) &dev_c, arr_size));
 
  for (int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
    c[i] = 1000 + i;
  }

  CUDA_WRAP ( hipMemcpy(dev_a, a, arr_size, hipMemcpyHostToDevice));  
  CUDA_WRAP ( hipMemcpy(dev_b, b, arr_size, hipMemcpyHostToDevice));
  CUDA_WRAP ( hipMemcpy(dev_c, c, arr_size, hipMemcpyHostToDevice));

  for (int i = 0; i < N; i++) {
    c[i] = 0;
  }

  add<<<NUM_BLOCKS,1>>>(dev_a, dev_b, dev_c);

  CUDA_WRAP( hipMemcpy(dev_c, c, arr_size, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < N; i++) {
    printf("%5d %5d %8d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
